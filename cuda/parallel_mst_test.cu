#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <limits.h>
#include "random_graph_generator.h"

#define BLOCK_SIZE 256

typedef long long int ll;
typedef pair<long long int, long long int> Edge; // Define edge type
typedef vector<vector<Edge>> Graph; // Define graph type

__global__ void initializeDistanceVector(ll* distanceVector, int numVertices, int source) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < numVertices) {
        distanceVector[tid] = (tid == source) ? 0 : LLONG_MAX;
    }
}

__global__ void updateDistanceVector(int numVertices, int* vertices, ll* weights, ll* distanceVector, bool* mstSet) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < numVertices && !mstSet[tid]) {
        for (int i = 0; i < vertices[tid]; ++i) {
            int v = vertices[tid * numVertices + i];
            ll w = weights[tid * numVertices + i];
            if (!mstSet[v] && w < distanceVector[v]) {
                distanceVector[v] = w;
            }
        }
    }
}

__global__ void findClosestNode(int numVertices, ll* distanceVector, bool* mstSet, int* closestNode) {
    __shared__ ll s_minDist[BLOCK_SIZE];
    __shared__ int s_minIdx[BLOCK_SIZE];

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    ll minDist = LLONG_MAX;
    int minIdx = -1;

    if(idx < numVertices && !mstSet[idx]) {
        minDist = distanceVector[idx];
        minIdx = idx;
    }

    s_minDist[tid] = minDist;
    s_minIdx[tid] = minIdx;

    __syncthreads();

    // Reduction to find minimum distance and its index
    for(int s = blockDim.x / 2; s > 0; s >>= 1) {
        if(tid < s && s_minDist[tid + s] < s_minDist[tid]) {
            s_minDist[tid] = s_minDist[tid + s];
            s_minIdx[tid] = s_minIdx[tid + s];
        }
        __syncthreads();
    }

    if(tid == 0) {
        closestNode[blockIdx.x] = s_minIdx[0];
    }
}

void printMST(const Graph& graph, bool* mstSet, int numVertices) {
    std::cout << "Minimum Spanning Tree (MST) Edges:\n";
    for(int i = 0; i < numVertices; ++i) {
        if(mstSet[i]) {
            for(const auto& edge : graph[i]) {
                if(mstSet[edge.first]) {
                    std::cout << "(" << i << " - " << edge.first << ") Weight: " << edge.second << "\n";
                }
            }
        }
    }
}

void printGraph(int n, const Graph& graph) {
    for (int u = 0; u < n; ++u) {
        for (const auto& neighbor : graph[u]) {
            int v = neighbor.first;
            long long int weight = neighbor.second;
            cout << u << " - " << v << " : " << weight << endl;
        }
    }
    printf("\n");
}

int main() {
    int numVertices = 10;
    int source = 0;
    Graph graph = generate(numVertices);

    printf("Generated graph:\n");
    printGraph(numVertices, graph);

    // Copy graph data to device
    int* d_vertices;
    ll* d_weights;
    hipMalloc((void**)&d_vertices, numVertices * numVertices * sizeof(int));
    hipMalloc((void**)&d_weights, numVertices * numVertices * sizeof(ll));

    for(int i = 0; i < numVertices; ++i) {
        for(int j = 0; j < graph[i].size(); ++j) {
            int v = graph[i][j].first;
            ll w = graph[i][j].second;
            hipMemcpy(&d_vertices[i * numVertices + j], &v, sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(&d_weights[i * numVertices + j], &w, sizeof(ll), hipMemcpyHostToDevice);
        }
    }

    // Allocate device memory
    ll* d_distanceVector;
    bool* d_mstSet;
    int* d_closestNode;
    hipMalloc((void**)&d_distanceVector, numVertices * sizeof(ll));
    hipMalloc((void**)&d_mstSet, numVertices * sizeof(bool));
    hipMalloc((void**)&d_closestNode, numVertices / BLOCK_SIZE * sizeof(int));

    // Initialize distance vector and MST set
    initializeDistanceVector<<<(numVertices + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_distanceVector, numVertices, source);
    hipMemset(d_mstSet, 0, numVertices * sizeof(bool));

    // Main loop for MST construction
    for (int i = 0; i < numVertices - 1; ++i) {
        findClosestNode<<<(numVertices + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(numVertices, d_distanceVector, d_mstSet, d_closestNode);

        int closestNode;
        hipMemcpy(&closestNode, d_closestNode, sizeof(int), hipMemcpyDeviceToHost);

        // Insert closestNode into MST
        hipMemset(&d_mstSet[closestNode], 1, sizeof(bool));

        updateDistanceVector<<<(numVertices + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(numVertices, d_vertices, d_weights, d_distanceVector, d_mstSet);
    }

    // Print the MST
    bool* mstSet = new bool[numVertices];
    hipMemcpy(mstSet, d_mstSet, numVertices * sizeof(bool), hipMemcpyDeviceToHost);
    printMST(graph, mstSet, numVertices);

    // Clean up
    delete[] mstSet;
    hipFree(d_distanceVector);
    hipFree(d_mstSet);
    hipFree(d_vertices);
    hipFree(d_weights);
    hipFree(d_closestNode);

    return 0;
}