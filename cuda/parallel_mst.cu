#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <vector>
#include <limits>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/scan.h>
#include <thrust/extrema.h>
#include "random_graph_generator.h"
#define INF INT_MAX
#define MIN_EDGE_WEIGHT 10
#define MAX_EDGE_WEIGHT 100
#define MAX_NODES 100000000
#define BLOCK_SIZE 4
#define NODES 8
using namespace std;
typedef pair<int, int> Edge; // Define edge type
typedef vector<vector<Edge>> Graph; // Define graph type

void adjacency_list_to_matrix(const vector<vector<pair<int, int>>>& adjList, int* adj_matrix, size_t n) {
    // Initialize the adjacency matrix with INF initially
    for (size_t i = 0; i < n; ++i) {
        for (size_t j = 0; j < n; ++j) {
            adj_matrix[i * n + j] = INF;
        }
    }

    // Populate the adjacency matrix with appropriate values from the adjacency list
    for (size_t i = 0; i < n; ++i) {
        for (const auto& edge : adjList[i]) {
            int vertex = edge.first;
            int weight = edge.second;
            adj_matrix[i * n + vertex] = weight;
        }
    }

    // Diagonal elements should be 0 (no self-loops)
    for (size_t i = 0; i < n; ++i) {
        adj_matrix[i * n + i] = 0;
    }
}

void print_adj_matrix(const int *adj_matrix) {
    for (size_t i = 0; i < NODES; ++i) {
        for (size_t j = 0; j < NODES; ++j) {
            int weight = adj_matrix[i * NODES + j];
            if (weight == INF) {
                cout << "INF\t";
            } else {
                cout << weight << "\t";
            }
        }
        cout << endl;
    }
}

//=========================================CUDA===============================================
void print_distance_vector(vector<int> distance_vector) {
    printf("distance vector: ");
    for(int i = 0 ; i < distance_vector.size(); i++){
        if(i == distance_vector.size() - 1){
            printf("%d", distance_vector[i]);
        } else {
            printf("%d, ", distance_vector[i]);
        }
    }
    printf("\n\n");
}

__global__ void local_closest_node(const int *d_distance_vector, int *d_min_weights, int *d_min_nodes,
                                 const bool *d_present_in_mst) {
    __shared__ int min_weight[NODES];
    __shared__ int closest_node[NODES];

    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory
    min_weight[tid] = d_distance_vector[index];
    closest_node[tid] = index;
    __syncthreads();

    // Ignore the elements that are already present in MST
    if (d_present_in_mst[index]) {
        min_weight[tid] = INF; // Set to maximum value to avoid selection
    }
    __syncthreads();

    // Reduction within the block
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            if (min_weight[tid + s] < min_weight[tid]) {
                min_weight[tid] = min_weight[tid + s];
                closest_node[tid] = closest_node[tid + s];
            }
        }
        __syncthreads();
    }

    // Store the minimum value and its index in global memory
    if (tid == 0) {
        d_min_weights[blockIdx.x] = min_weight[0];
        d_min_nodes[blockIdx.x] = closest_node[0];
        //printf("local - min w %d, min n %d\n", min_weight[0], closest_node[0]);
    }
}

__global__ void update_distances(
        const int *d_matrix, int *d_mst, int *d_distance_vector, int final_min_node, const bool *d_present_in_mst) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < NODES && tid != final_min_node && !d_present_in_mst[tid]) {
        int index = final_min_node * NODES + tid; // Assuming d_matrix is a flattened 2D array
        if (d_matrix[index] < d_distance_vector[tid]) {
            d_mst[tid] = final_min_node;
            d_distance_vector[tid] = d_matrix[index];
        }
    }
}

int main() {
    //HOST MEMORY
    int numBlocks = (NODES + BLOCK_SIZE - 1) / BLOCK_SIZE;

    int source = 0;
    Graph graph = generate(NODES);
    int edges = tot_edges(graph, NODES);
    printf("Generated graph of %d vertices and %d edges:\n", NODES, edges);
    print_graph(graph, NODES);

    vector<int> distance_vector(NODES, INF);
    distance_vector[source] = 0;

    vector<int> mst(NODES, -1);
    bool present_in_mst[NODES] = { false };

    int *adj_matrix = new int[NODES * NODES];
    adjacency_list_to_matrix(graph, adj_matrix, NODES);
    //print_adj_matrix(adj_matrix);

    //DEVICE MEMORY
    int* d_matrix;      //partitioned adjacency matrix
    hipMalloc((void **)&d_matrix, NODES * NODES * sizeof(int));
    hipMemcpy(d_matrix, adj_matrix, NODES * NODES * sizeof(int), hipMemcpyHostToDevice);

    int* d_distance_vector; //vector distance: distance from the MST to each node
    hipMalloc((void **)&d_distance_vector, NODES * sizeof(int));
    hipMemcpy(d_distance_vector, distance_vector.data(), NODES * sizeof(int), hipMemcpyHostToDevice);

    bool* d_present_in_mst;
    hipMalloc((void **)&d_present_in_mst, NODES * sizeof(int));

    int* d_min_weights;
    hipMalloc((void **)&d_min_weights, BLOCK_SIZE * sizeof (int));

    int* d_min_nodes;
    hipMalloc((void **)&d_min_nodes, BLOCK_SIZE * sizeof (int));

    int *d_mst;
    hipMalloc((void **)&d_mst, NODES * sizeof(int));

    int nBytes = NODES * sizeof(int) + NODES * sizeof(int);
    for (int i = 0; i < NODES; i++) {
        //printf("\n===== STEP NUMBER %d ======\n", i + 1);
        //print_distance_vector(distance_vector);

        // Launch kernel with appropriate block and thread configuration
        local_closest_node<<<numBlocks, BLOCK_SIZE, nBytes>>>(
                d_distance_vector, d_min_weights, d_min_nodes, d_present_in_mst);
        hipDeviceSynchronize();        // Wait for kernel to finish

        //Find global closest node from the local solutions
        thrust::device_ptr<int> thrust_weights_ptr(d_min_weights);
        thrust::device_ptr<int> thrust_nodes_ptr = thrust::device_pointer_cast(d_min_nodes);
        thrust::device_ptr<int> min_ptr = thrust::min_element(
                thrust::device, thrust_weights_ptr, thrust_weights_ptr + numBlocks);

        int final_min_weight = *min_ptr;
        int final_min_node = thrust_nodes_ptr[min_ptr - thrust_weights_ptr];
        present_in_mst[final_min_node] = true;
        hipMemcpy(d_present_in_mst, present_in_mst, NODES * sizeof(bool), hipMemcpyHostToDevice);

/*
        printf("Minimum weight: %d, node: %d\n", final_min_weight, final_min_node);
        for(int j = 0 ; j < NODES ; j++) {
            if(present_in_mst[j]){
                printf("Node %d is present in MST\n", j);
            }
        }
*/
        update_distances<<<numBlocks, BLOCK_SIZE>>>(
                d_matrix, d_mst, d_distance_vector, final_min_node, d_present_in_mst);

        hipDeviceSynchronize();

        hipMemcpy(distance_vector.data(), d_distance_vector, NODES * sizeof(int), hipMemcpyDeviceToHost);
    }

    hipMemcpy(mst.data(), d_mst, NODES * sizeof(int), hipMemcpyDeviceToHost);

    // Construct MST graph from mst array
    Graph mstGraph(NODES);
    for (int i = 1 ; i < NODES; ++i) {
        int u = mst[i];
        mstGraph[u].emplace_back(i, distance_vector[i]);
    }
    printf("\nThe MST is:\n");
    print_graph(mstGraph, NODES);

    // Free memory
    delete[] adj_matrix;
    hipFree(d_matrix);
    hipFree(d_distance_vector);
    hipFree(d_present_in_mst);
    hipFree(d_min_weights);
    hipFree(d_min_nodes);
    hipFree(d_mst);
    return 0;
}