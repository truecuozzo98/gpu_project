#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <vector>
#include <set>
#include <limits>
#define INF INT_MAX
#define MIN_EDGE_WEIGHT 10
#define MAX_EDGE_WEIGHT 100
#define MAX_NODES 100000000
#define BLOCK_SIZE 2
#define NODES 4
using namespace std;
typedef pair<int, int> Edge; // Define edge type
typedef vector<vector<Edge>> Graph; // Define graph type

void printGraph(const Graph& graph) {
    for (int u = 0; u < NODES; ++u) {
        for (const auto& neighbor : graph[u]) {
            int v = neighbor.first;
            int weight = neighbor.second;
            cout << u << " - " << v << " : " << weight << endl;
        }
    }
    printf("\n");
}

Graph generate() {
    // Use a random device to seed the random number engine
    random_device rd;
    // Use the Mersenne Twister engine for randomness
    mt19937 mt(rd());
    // Define the distribution for integers
    uniform_int_distribution<int> random_weight(MIN_EDGE_WEIGHT, MAX_EDGE_WEIGHT);
    uniform_int_distribution<int> random_extra_edges(1, ((NODES - 1) * NODES)/2 - (NODES-1));
    uniform_int_distribution<int> random_node(0, NODES-1);

    Graph adjacency(NODES);

    int extra_edges = random_extra_edges(mt);

    if(NODES - 1 + extra_edges > MAX_NODES){
        int difference = MAX_NODES - (NODES - 1);
        uniform_int_distribution<int> random_difference(0, difference +1);
        extra_edges = random_difference(mt);
    }

    vector<int> graph(NODES);

    for(int i = 0; i < NODES; ++i){
        graph[i] = i;
    }

    shuffle(graph.begin(),graph.end(), mt19937(random_device()()));

    set<Edge> present_edge;

    for(int i = 1; i < NODES; ++i){
        uniform_int_distribution<int> random_add(0, i - 1);
        int add = random_add(mt);
        int weight = random_weight(mt);
        adjacency[graph[i]].emplace_back(graph[add], weight);
        adjacency[graph[add]].emplace_back(graph[i], weight);
        present_edge.insert(make_pair(min(graph[add], graph[i]), max(graph[add], graph[i])));
    }

    for(int i = 1; i <= extra_edges; ++i){
        int weight = random_weight(mt);
        while(true){
            int node1 = random_node(mt);
            int node2 = random_node(mt);
            if(node1 == node2) continue;
            if(present_edge.find(make_pair(min(node1, node2), max(node1, node2))) == present_edge.end()){
                adjacency[node1].emplace_back(node2, weight);
                adjacency[node2].emplace_back(node1, weight);
                present_edge.insert(make_pair(min(node1, node2), max(node1, node2)));
                break;
            }
        }
    }
    return adjacency;
}

int totEdges(int n, const Graph& graph) {
    int count = 0;
    for (int u = 0; u < n; ++u) {
        count += graph[u].size();
    }
    return count;
}

void adjacencyListToMatrix(const vector<vector<pair<int, int>>>& adjList, int* adjMatrix, size_t n) {
    // Initialize the adjacency matrix with INF initially
    for (size_t i = 0; i < n; ++i) {
        for (size_t j = 0; j < n; ++j) {
            adjMatrix[i * n + j] = INF;
        }
    }

    // Populate the adjacency matrix with appropriate values from the adjacency list
    for (size_t i = 0; i < n; ++i) {
        for (const auto& edge : adjList[i]) {
            int vertex = edge.first;
            int weight = edge.second;
            adjMatrix[i * n + vertex] = weight;
        }
    }

    // Diagonal elements should be 0 (no self-loops)
    for (size_t i = 0; i < n; ++i) {
        adjMatrix[i * n + i] = 0;
    }
}

void printAdjMatrix(const int *adjMatrix) {
    for (size_t i = 0; i < NODES; ++i) {
        for (size_t j = 0; j < NODES; ++j) {
            int weight = adjMatrix[i * NODES + j];
            if (weight == INF) {
                cout << "INF\t";
            } else {
                cout << weight << "\t";
            }
        }
        cout << endl;
    }
}

//=========================================CUDA===============================================
void printDistanceVector(const char *s, vector<int> distanceVector) {
    printf("%s", (const char *const) s);
    for(int i = 0 ; i < distanceVector.size(); i++){
        if(i == distanceVector.size() - 1){
            printf("%d", distanceVector[i]);
        } else {
            printf("%d, ", distanceVector[i]);
        }
    }
    printf("\n\n");
}


__global__ void localClosestNode(const int *d_distanceVector, int *d_minWeights, int *d_minNodes,
                                 const bool *d_presentInMST) {
    __shared__ int minWeight[NODES];
    __shared__ int smallestNodeIndex[NODES];

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory
    minWeight[tid] = d_distanceVector[index];
    smallestNodeIndex[tid] = index;
    __syncthreads();

    /*for (int i = 0 ; i<NODES; i++) {
        printf("node: %d present: %d\n", i, d_presentInMST[i]);
    }*/

    // Ignore the elements that are already present in MST
    if (d_presentInMST[index]) {
        minWeight[tid] = INF; // Set to maximum value to avoid selection
    }
    __syncthreads();

    // Reduction within the block
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            printf("block %d thread %d tid+s %d index %d\n", bid, tid, tid+s, index);
            if (minWeight[tid + s] < minWeight[tid]) {
                minWeight[tid] = minWeight[tid + s];
                smallestNodeIndex[tid] = smallestNodeIndex[tid + s];
            }
        }
        __syncthreads();
    }

    // Store the minimum value and its index in global memory
    if (tid == 0) {
        d_minWeights[blockIdx.x] = minWeight[0];
        d_minNodes[blockIdx.x] = smallestNodeIndex[0];
        printf("local - block: %d, min_node: %d, min_weight: %d\n", bid, smallestNodeIndex[0], minWeight[0]);
    }
}


__global__ void globalClosestNode(int *min_val, int *min_index, bool *d_presentInMST/*, vector<int> d_mst*/) {
    __shared__ int s_minIndex[NODES];
    __shared__ int s_minValue[NODES];

    int tid = threadIdx.x;

    // Load data into shared memory
    s_minIndex[tid] = min_index[tid];
    s_minValue[tid] = min_val[tid];

    __syncthreads();

    for (int s = NODES / 2; s > 0; s /= 2) {
        if (tid < s) {
            if (s_minValue[tid + s] < s_minValue[tid] && s_minValue[tid+s] != 0) {
                s_minValue[tid] = s_minValue[tid + s];
                s_minIndex[tid] = s_minIndex[tid + s /*+ (threadIdx.x * blockDim.x)*/];
            }
        }
        __syncthreads();
    }

    if (tid == 0) {
        min_val[0] = s_minValue[0];
        min_index[0] = s_minIndex[0];
        //d_presentInMST[s_minValue[0]] = true;
    }
}

int main() {
    int source = 0;
    Graph graph = generate();
    int edges = totEdges(NODES, graph);

    vector<int> distanceVector(NODES, INF); // Key values used to pick minimum weight edge in cut
    distanceVector[source] = 0;
    vector<int> mst(NODES); // Array to store constructed MST parent, MST has numVertices-1 edges
    auto *presentInMST = new bool[NODES]; // To represent set of vertices not yet included in MST
    for (int i = 0; i < NODES; i++) {
        if(i == source) {
            presentInMST[i] = true;
        }
        presentInMST[i] = false;
    }

    printf("Generated graph of %d vertices and %d edges:\n", NODES, edges);
    printGraph(graph);

    auto* adjMatrix = new int[NODES * NODES];
    adjacencyListToMatrix(graph, adjMatrix, NODES);
    printAdjMatrix(adjMatrix);

    // Allocate memory on device
    int numBlocks = (NODES + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int* d_matrix; //partitioned adjacency matrix
    size_t matrixSize = NODES * NODES * sizeof(int);
    hipMalloc((void **)&d_matrix, matrixSize);

    int* d_distanceVector; //vector distance: distance from the MST to each node
    size_t nodeSize = NODES * sizeof(int);
    hipMalloc((void **)&d_distanceVector, nodeSize);
    hipMemcpy(d_distanceVector, distanceVector.data(), nodeSize, hipMemcpyHostToDevice);

    bool* d_presentInMST;
    hipMalloc((void **)&d_presentInMST, nodeSize);

    int* d_minWeights;
    hipMalloc((void **)&d_minWeights, BLOCK_SIZE * sizeof (int));

    int* d_minNodes;
    hipMalloc((void **)&d_minNodes, BLOCK_SIZE * sizeof (int));

    int *d_minIndex;
    hipMalloc((void**)&d_minIndex, sizeof(int));

    vector<int> d_mst(NODES);
    hipMalloc((void **)&d_mst, nodeSize);

    // Copy data from host to device
    hipMemcpy(d_matrix, adjMatrix, matrixSize, hipMemcpyHostToDevice);

    for (int i = 0; i < NODES - 1; i++) {
        printf("\n===== STEP NUMBER %d ======\n", i+1);
        printDistanceVector("distance vector pre update: ", distanceVector);

        // Launch kernel with appropriate block and thread configuration
        localClosestNode<<<numBlocks, BLOCK_SIZE>>>(d_distanceVector, d_minWeights, d_minNodes, d_presentInMST);
        // Wait for kernel to finish
        hipDeviceSynchronize();

        globalClosestNode<<<1, BLOCK_SIZE>>>(d_minWeights, d_minNodes, d_presentInMST);
        hipDeviceSynchronize();

        int final_min_val, final_min_index;
        hipMemcpy(&final_min_val, d_minWeights, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&final_min_index, d_minNodes, sizeof(int), hipMemcpyDeviceToHost);
        //hipMemcpy(presentInMST, d_presentInMST, NODES * sizeof(int), hipMemcpyDeviceToHost);

        printf("Minimum value: %d, Index: %d\n", final_min_val, final_min_index);
        presentInMST[final_min_index] = true;
        for(int j = 0 ; j < NODES ; j++) {
            if(presentInMST[j]){
                printf("Node %d is present in MST\n", j);
            }
        }

        hipMemcpy(d_presentInMST, presentInMST, NODES * sizeof(int), hipMemcpyHostToDevice);

        /*
        // Launch kernel to find index of minimum value
        globalClosestNode<<<numBlocks, BLOCK_SIZE>>>(
                d_distanceVector, NODES, d_minIndex, d_presentInMST, d_mst);
        hipDeviceSynchronize();

        int final_min_val, final_min_index;
        hipMemcpy(&final_min_val, d_minWeights, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&final_min_index, d_minNodes, sizeof(int), hipMemcpyDeviceToHost);
        */

        /*
        printf("Minimum value: %d, Index: %d\n", final_min_val, final_min_index);

        // Copy result back to host
        int minIndex = -1;
        hipMemcpy(&minIndex, d_minIndex, sizeof(int), hipMemcpyDeviceToHost);
        presentInMST[minIndex] = true;
        hipMemcpy(d_presentInMST, presentInMST, NODES * sizeof(int), hipMemcpyHostToDevice);

        printf("Index of minimum value: %d\n", minIndex);
        for(int j = 0 ; j < NODES ; j++) {
            if(presentInMST[j]){
                printf("Node %d is present in MST\n", j);
            }
        }*/

        //TODO: parallel version
        for (const Edge& neighbor : graph[final_min_index]) {
            int v = neighbor.first;
            int weight = neighbor.second;
            if (!presentInMST[v] && weight < distanceVector[v]) {
                mst[v] = final_min_index;
                distanceVector[v] = weight;
            }
        }
        hipMemcpy(d_distanceVector, distanceVector.data(), NODES * sizeof(int), hipMemcpyHostToDevice);

        printDistanceVector("distance vector post update: ", distanceVector);
    }

    // Construct MST graph from mst array
    Graph mstGraph(NODES);
    for (int i = 0 ; i < NODES; ++i) {
        int u = mst[i];
        mstGraph[u].emplace_back(i, distanceVector[i]);
    }
    printGraph(mstGraph);

    /*
    printf("Index of minimum value: %d\n", minIndex);
    for(int i = 0 ; i < NODES ; i++) {
        if(presentInMST[i]){
            printf("Node %d is present in MST\n", i);
        }
    }*/

    // Free memory
    free(adjMatrix);
    free(presentInMST);
    //hipFree(d_result);
    hipFree(d_matrix);

    return 0;
}